#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__host__ __device__ int outInvariant(int inValue) {
  return inValue * inValue;
}

__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}

__global__ void s2g_gpu_scatter_kernel(int *in, int *out, int len) {
	int inIdx = blockIdx.x; 	
	int intermediate = outInvariant(in[inIdx]);
	for (int outIdx = 0; outIdx < len; ++outIdx) {
		int val = outDependent(intermediate, inIdx, outIdx);
		atomicAdd(&out[outIdx], val);
	}
}



__global__ void s2g_gpu_gather_kernel(int *in, int *out, int len) {
  
	int outIdx = blockIdx.x;
	for (int inIdx = 0; inIdx < len; ++inIdx){
		int intermediate = outInvariant(in[inIdx]);
		int val = outDependent(intermediate, inIdx, outIdx);
		atomicAdd(&out[outIdx], val);
	}
}




static void s2g_cpu_scatter(int *in, int *out, int len) {
  for (int inIdx = 0; inIdx < len; ++inIdx) {
    int intermediate = outInvariant(in[inIdx]);
    for (int outIdx = 0; outIdx < len; ++outIdx) {
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}

static void s2g_cpu_gather(int *in, int *out, int len) {
	for (int outIdx = 0; outIdx < len; ++outIdx){
		for (int inIdx = 0; inIdx < len; ++inIdx){
			int intermediate = outInvariant(in[inIdx]);
			out[outIdx] += outDependent(intermediate, inIdx, outIdx);
		}
	}	
}

static void s2g_gpu_scatter(int *in, int *out, int len) {
	//int* dev_in;
	//int* dev_out;
	//hipMalloc( (void**)&dev_in, len*sizeof(int) );
	//hipMemcpy( dev_in, in, len*sizeof(int), hipMemcpyHostToDevice );

	//hipMalloc( (void**)&dev_out, len*sizeof(int) );
	//hipMemcpy( dev_out, out, len*sizeof(int), hipMemcpyHostToDevice );

	s2g_gpu_scatter_kernel<<<len,1>>>(in, out, len);
	//hipMemcpy( out, dev_out, len*sizeof(int), hipMemcpyDeviceToHost );

	//hipFree(dev_in);
	//hipFree(dev_out);
}

static void s2g_gpu_gather(int *in, int *out, int len) {
 
	//int* dev_in;
	//int* dev_out;
	//hipMalloc( (void**)&dev_in, len*sizeof(int) );
	//hipMemcpy( dev_in, in, len*sizeof(int), hipMemcpyHostToDevice );

	//hipMalloc( (void**)&dev_out, len*sizeof(int) );
	//hipMemcpy( dev_out, out, len*sizeof(int), hipMemcpyHostToDevice );

	s2g_gpu_gather_kernel<<<len,1>>>(in, out, len);
	//hipMemcpy( out, dev_out, len*sizeof(int), hipMemcpyDeviceToHost );

	//hipFree(dev_in);
	//hipFree(dev_out);
	
	
}




int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  int *hostInput;
  int *hostOutput;
  int *deviceInput;
  int *deviceOutput;
  size_t byteCount;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &inputLength,
                              "Integer");
  hostOutput = (int *)malloc(inputLength * sizeof(int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  byteCount = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, byteCount));
  wbCheck(hipMalloc((void **)&deviceOutput, byteCount));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, byteCount,
                     hipMemcpyHostToDevice));
  wbCheck(hipMemset(deviceOutput, 0, byteCount));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //////////////////////////////////////////
  // CPU Scatter Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing CPU Scatter computation");
  s2g_cpu_scatter(hostInput, hostOutput, inputLength);
  wbTime_stop(Compute, "Performing CPU Scatter computation");
  wbSolution(args, hostOutput, inputLength);
  memset(hostOutput, 0, byteCount);

  //////////////////////////////////////////
  // GPU Scatter Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Scatter computation");
  s2g_gpu_scatter(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Scatter computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);
  wbCheck(hipMemset(deviceOutput, 0, byteCount));

  //////////////////////////////////////////
  // CPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing CPU Gather computation");
  s2g_cpu_gather(hostInput, hostOutput, inputLength);
  wbTime_stop(Compute, "Performing CPU Gather computation");
  wbSolution(args, hostOutput, inputLength);
  memset(hostOutput, 0, byteCount);

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Gather computation");
  s2g_gpu_gather(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Gather computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);
  wbCheck(hipMemset(deviceOutput, 0, byteCount));

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostInput);
  free(hostOutput);

  return 0;
}
